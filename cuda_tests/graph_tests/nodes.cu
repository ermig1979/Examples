#include "hip/hip_runtime.h"
#include "nodes.h"

__global__ void add_kernel(int size, const float* a, const float* b, float* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) 
    {
        c[i] = a[i] + b[i];
    }
}

//__global__ void add_kernel_cublas(hipblasHandle_t handle, int size, const float* a, const float* b, float* c)
//{
//    //ublasHandle_t handle;
//    //hipblasCreate(&handle);
//
//    hipblasScopy(handle, size, a, 1, c, 1);
//
//    const float alpha = 1.0f;
//    hipblasSaxpy(handle, size, &alpha, b, 1, c, 1);
////
//    //hipblasDestroy(handle);
//}
