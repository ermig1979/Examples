#include "hip/hip_runtime.h"
#include "defs.h"

__global__ void gemm_v0a(int M, int N, int K, const float * A, const float * B, float * C)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < M && j < N)
    {
        C[i * N + j] = 0;
        for (int k = 0; k < K; ++k)
            C[i * N + j] += A[i * K + k] * B[k * N + j];
    }
}

int gemm_gpu_v0a(int M, int N, int K, const float * A, const float * B, float * C)
{
    const int n = repeats(M, N, K, 0.03);
    const int TS = 16;
    dim3 grid(TS, TS);
    dim3 block((N + TS - 1)/TS, (M + TS - 1)/TS);
    for (int i = 0; i < n; ++i)
        gemm_v0a<<<block, grid>>>(M, N, K, A, B, C);
    assert(hipGetLastError() == hipSuccess);
    return n;
}