#include "hip/hip_runtime.h"
#include "defs.h"

#define S 32

__global__ void gemm_v2(int M, int N, int K, const float * A, const float * B, float * C)
{
    int Ma = M / S * S;
    int Na = N / S * S;
    int Ka = K / S * S;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < M && j < N)
    {
        int kB = 0;
        float c = 0;
        if(i < Ma && j < Na)
        {
            for (; kB < Ka; kB += S) 
            {
                __shared__ float sA[S][S];
                __shared__ float sB[S][S];
                sA[threadIdx.y][threadIdx.x] = A[i * K + (kB + threadIdx.x)];
                sB[threadIdx.y][threadIdx.x] = B[(kB + threadIdx.y) * N + j];
                __syncthreads();
                for (int k = 0; k < S; ++k)
                    c += sA[threadIdx.y][k] * sB[k][threadIdx.x];
                __syncthreads();
            }
        }
        for (int k = kB; k < K; ++k)
            c += A[i * K + k] * B[k * N + j];
        C[i * N + j] = c;
    }
}

int gemm_gpu_v2(int M, int N, int K, const float * A, const float * B, float * C)
{
    dim3 grid(S, S);
    dim3 block((N + S - 1)/S, (M + S - 1)/S);
    const int n = repeats(M, N, K, 0.170);
    for (int i = 0; i < n; ++i)
        gemm_v2<<<block, grid>>>(M, N, K, A, B, C);
    assert(hipGetLastError() == hipSuccess);
    return n;
}